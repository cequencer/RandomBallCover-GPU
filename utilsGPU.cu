/* This file is part of the Random Ball Cover (RBC) library.
 * (C) Copyright 2010, Lawrence Cayton [lcayton@tuebingen.mpg.de]
 */

#ifndef UTILSGPU_CU
#define UTILSGPU_CU

#include<hip/hip_runtime.h>
#include<stdio.h>
#include "defs.h"
#include "utilsGPU.h"

void copyAndMove(matrix *dx, const matrix *x){
  dx->r = x->r; 
  dx->c = x->c;
  dx->pr = x->pr;
  dx->pc = x->pc;
  dx->ld = x->ld;

  checkErr( hipMalloc( (void**)&(dx->mat), dx->pr*dx->pc*sizeof(*(dx->mat)) ) );
  hipMemcpy( dx->mat, x->mat, dx->pr*dx->pc*sizeof(*(dx->mat)), hipMemcpyHostToDevice );
  
}


void copyAndMoveI(intMatrix *dx, const intMatrix *x){
  dx->r = x->r; 
  dx->c = x->c;
  dx->pr = x->pr;
  dx->pc = x->pc;
  dx->ld = x->ld;

  checkErr( hipMalloc( (void**)&(dx->mat), dx->pr*dx->pc*sizeof(*(dx->mat)) ) );
  hipMemcpy( dx->mat, x->mat, dx->pr*dx->pc*sizeof(*(dx->mat)), hipMemcpyHostToDevice );
  
}


void copyAndMoveC(charMatrix *dx, const charMatrix *x){
  dx->r = x->r; 
  dx->c = x->c;
  dx->pr = x->pr;
  dx->pc = x->pc;
  dx->ld = x->ld;

  checkErr( hipMalloc( (void**)&(dx->mat), dx->pr*dx->pc*sizeof(*(dx->mat)) ) );
  hipMemcpy( dx->mat, x->mat, dx->pr*dx->pc*sizeof(*(dx->mat)), hipMemcpyHostToDevice );
  
}


void checkErr(hipError_t cError){
  if(cError != hipSuccess){
    fprintf(stderr,"GPU-related error:\n\t%s \n", hipGetErrorString(cError) );
    fprintf(stderr,"exiting ..\n");
    exit(1);
  }
  
}

void checkErr(char* loc, hipError_t cError){
  printf("in %s:\n",loc);
  checkErr(cError);
}

#endif
