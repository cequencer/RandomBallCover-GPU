#include "hip/hip_runtime.h"
/* This file is part of the Random Ball Cover (RBC) library.
 * (C) Copyright 2010, Lawrence Cayton [lcayton@tuebingen.mpg.de]
 */

#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<sys/time.h>
#include<math.h>
#include "defs.h"
#include "utils.h"
#include "utilsGPU.h"
#include "rbc.h"
#include "brute.h"
#include "sKernel.h"

void parseInput(int,char**);
void readData(char*,matrix);
void readDataText(char*,matrix);
void evalNNerror(matrix, matrix, unint*);
void evalKNNerror(matrix,matrix,intMatrix);

char *dataFileX, *dataFileQ, *outFile;
char runBrute=0, runEval=0;
unint n=0, m=0, d=0, numReps=0;
unint deviceNum=0;
int main(int argc, char**argv){
  matrix x, q;
  intMatrix nnsRBC;
  matrix distsRBC;
  struct timeval tvB,tvE;
  hipError_t cE;
  rbcStruct rbcS;

  printf("*****************\n");
  printf("RANDOM BALL COVER\n");
  printf("*****************\n");
  
  parseInput(argc,argv);
  
  printf("Using GPU #%d\n",deviceNum);
  if(hipSetDevice(deviceNum) != hipSuccess){
    printf("Unable to select device %d.. exiting. \n",deviceNum);
    exit(1);
  }
  
  size_t memFree, memTot;
  hipMemGetInfo(&memFree, &memTot);
  printf("GPU memory free = %lu/%lu (MB) \n",(unsigned long)memFree/(1024*1024),(unsigned long)memTot/(1024*1024));

  initMat( &x, n, d );
  initMat( &q, m, d );
  x.mat = (real*)calloc( sizeOfMat(x), sizeof(*(x.mat)) );
  q.mat = (real*)calloc( sizeOfMat(q), sizeof(*(q.mat)) );
    
  //Load data 
  readData( dataFileX, x );
  readData( dataFileQ, q );

  //Allocate space for NNs and dists
  initIntMat( &nnsRBC, m, K );
  initMat( &distsRBC, m, K );
  nnsRBC.mat = (unint*)calloc( sizeOfIntMat(nnsRBC), sizeof(*nnsRBC.mat) );
  distsRBC.mat = (real*)calloc( sizeOfMat(distsRBC), sizeof(*distsRBC.mat) );

  printf("\nrunning rbc..\n");
  //Build the RBC
  gettimeofday(&tvB,NULL);
  buildRBC(x, &rbcS, numReps, numReps);
  gettimeofday(&tvE,NULL);
  printf("\t.. build time for rbc = %6.4f \n",timeDiff(tvB,tvE));
  
  //This finds the 32-NNs; if you are only interested in the 1-NN, use queryRBC(..) instead
  gettimeofday(&tvB,NULL);
  kqueryRBC(q, rbcS, nnsRBC, distsRBC);
  gettimeofday(&tvE,NULL);
  printf("\t.. query time for krbc = %6.4f \n",timeDiff(tvB,tvE));
  
  if( runBrute ){
    intMatrix nnsBrute;
    matrix distsBrute;
    initIntMat( &nnsBrute, m, K );
    nnsBrute.mat = (unint*)calloc( sizeOfIntMat(nnsBrute), sizeof(*nnsBrute.mat) );
    initMat( &distsBrute, m, K );
    distsBrute.mat = (real*)calloc( sizeOfMat(distsBrute), sizeof(*distsBrute.mat) );
    
    printf("running k-brute force..\n");
    gettimeofday(&tvB,NULL);
    bruteK(x,q,nnsBrute,distsBrute);
    gettimeofday(&tvE,NULL);
    printf("\t.. time elapsed = %6.4f \n",timeDiff(tvB,tvE));
    
    free(nnsBrute.mat);
    free(distsBrute.mat);
  }

  cE = hipGetLastError();
  if( cE != hipSuccess ){
    printf("Execution failed; error type: %s \n", hipGetErrorString(cE) );
  }
  
  if( runEval )
    evalKNNerror(x,q,nnsRBC);
  
  destroyRBC(&rbcS);
  hipDeviceReset();
  free(nnsRBC.mat);
  free(distsRBC.mat);
  free(x.mat);
  free(q.mat);
}


void parseInput(int argc, char **argv){
  int i=1;
  if(argc <= 1){
    printf("\nusage: \n  testRBC -x datafileX -q datafileQ  -n numPts (DB) -m numQueries -d dim -r numReps [-o outFile] [-g GPU num] [-b] [-e]\n\n");
    printf("\tdatafileX    = binary file containing the database\n");
    printf("\tdatafileQ    = binary file containing the queries\n");
    printf("\tnumPts       = size of database\n");
    printf("\tnumQueries   = number of queries\n");
    printf("\tdim          = dimensionailty\n");
    printf("\tnumReps      = number of representatives\n");
    printf("\toutFile      = output file (optional); stored in text format\n");
    printf("\tGPU num      = ID # of the GPU to use (optional) for multi-GPU machines\n");
    printf("\n\tuse -b to run brute force in addition the RBC\n");
    printf("\tuse -e option to run evaluation routine\n");
    printf("\n\n");
    exit(0);
  }
  
  while(i<argc){
    if(!strcmp(argv[i], "-x"))
      dataFileX = argv[++i];
    else if(!strcmp(argv[i], "-q"))
      dataFileQ = argv[++i];
    else if(!strcmp(argv[i], "-n"))
      n = atoi(argv[++i]);
    else if(!strcmp(argv[i], "-m"))
      m = atoi(argv[++i]);
    else if(!strcmp(argv[i], "-d"))
      d = atoi(argv[++i]);
    else if(!strcmp(argv[i], "-r"))
      numReps = atoi(argv[++i]);
    else if(!strcmp(argv[i], "-o"))
      outFile = argv[++i];
    else if(!strcmp(argv[i], "-g"))
      deviceNum = atoi(argv[++i]);
    else{
      fprintf(stderr,"%s : unrecognized option.. exiting\n",argv[i]);
      exit(1);
    }
    i++;
  }

  if( !n || !m || !d || !numReps || !dataFileX || !dataFileQ ){
    fprintf(stderr,"more arguments needed.. exiting\n");
    exit(1);
  }
  
  if(numReps>n){
    fprintf(stderr,"can't have more representatives than points.. exiting\n");
    exit(1);
  }
}


void readData(char *dataFile, matrix x){
  unint i;
  FILE *fp;
  unint numRead;

  fp = fopen(dataFile,"r");
  if(fp==NULL){
    fprintf(stderr,"error opening file.. exiting\n");
    exit(1);
  }
    
  for( i=0; i<x.r; i++ ){ //can't load everything in one fread
                           //because matrix is padded.
    numRead = fread( &x.mat[IDX( i, 0, x.ld )], sizeof(real), x.c, fp );
    if(numRead != x.c){
      fprintf(stderr,"error reading file.. exiting \n");
      exit(1);
    }
  }
  fclose(fp);
}


void readDataText(char *dataFile, matrix x){
  FILE *fp;
  real t;
  int i,j;

  fp = fopen(dataFile,"r");
  if(fp==NULL){
    fprintf(stderr,"error opening file.. exiting\n");
    exit(1);
  }
    
  for(i=0; i<x.r; i++){
    for(j=0; j<x.c; j++){
      if(fscanf(fp,"%f ", &t)==EOF){
	fprintf(stderr,"error reading file.. exiting \n");
	exit(1);
      }
      x.mat[IDX( i, j, x.ld )]=(real)t;
    }
  }
  fclose(fp);
}


//find the error rate of a set of NNs, then print it.
void evalNNerror(matrix x, matrix q, unint *NNs){
  struct timeval tvB, tvE;
  unint i;

  printf("\nComputing error rates (this might take a while)\n");
  real *ranges = (real*)calloc(q.pr,sizeof(*ranges));
  for(i=0;i<q.r;i++){
    if(NNs[i]>n) printf("error");
    ranges[i] = distVec(q,x,i,NNs[i]) - 10e-6;
  }

  unint *cnts = (unint*)calloc(q.pr,sizeof(*cnts));
  gettimeofday(&tvB,NULL);
  bruteRangeCount(x,q,ranges,cnts);
  gettimeofday(&tvE,NULL);
  
  long int nc=0;
  for(i=0;i<m;i++){
    nc += cnts[i];
  }
  double mean = ((double)nc)/((double)m);
  double var = 0.0;
  for(i=0;i<m;i++) {
    var += (((double)cnts[i])-mean)*(((double)cnts[i])-mean)/((double)m);
  }
  printf("\tavg rank = %6.4f; std dev = %6.4f \n\n", mean, sqrt(var));
  printf("(range count took %6.4f) \n", timeDiff(tvB, tvE));
  
  if(outFile){
    FILE* fp = fopen(outFile, "a");
    fprintf( fp, "%d %6.5f %6.5f \n", numReps, mean, sqrt(var) );
    fclose(fp);
  }

  free(ranges);
  free(cnts);
}


//evals the error rate of k-nns
void evalKNNerror(matrix x, matrix q, intMatrix NNs){
  struct timeval tvB, tvE;
  unint i,j,k;

  unint m = q.r;
  printf("\nComputing error rates (this might take a while)\n");
  
  unint *ol = (unint*)calloc( q.r, sizeof(*ol) );
  
  intMatrix NNsB;
  NNsB.r=q.r; NNsB.pr=q.pr; NNsB.c=NNsB.pc=32; NNsB.ld=NNsB.pc;
  NNsB.mat = (unint*)calloc( NNsB.pr*NNsB.pc, sizeof(*NNsB.mat) );
  matrix distsBrute;
  distsBrute.r=q.r; distsBrute.pr=q.pr; distsBrute.c=distsBrute.pc=K; distsBrute.ld=distsBrute.pc;
  distsBrute.mat = (real*)calloc( distsBrute.pr*distsBrute.pc, sizeof(*distsBrute.mat) );

  gettimeofday(&tvB,NULL);
  bruteK(x,q,NNsB,distsBrute);
  gettimeofday(&tvE,NULL);

   //calc overlap
  for(i=0; i<m; i++){
    for(j=0; j<K; j++){
      for(k=0; k<K; k++){
	ol[i] += ( NNs.mat[IDX(i, j, NNs.ld)] == NNsB.mat[IDX(i, k, NNsB.ld)] );
      }
    }
  }

  long int nc=0;
  for(i=0;i<m;i++){
    nc += ol[i];
  }

  double mean = ((double)nc)/((double)m);
  double var = 0.0;
  for(i=0;i<m;i++) {
    var += (((double)ol[i])-mean)*(((double)ol[i])-mean)/((double)m);
  }
  printf("\tavg overlap = %6.4f/%d; std dev = %6.4f \n", mean, K, sqrt(var));

  FILE* fp;
  if(outFile){
    fp = fopen(outFile, "a");
    fprintf( fp, "%d %6.5f %6.5f ", numReps, mean, sqrt(var) );
  }

  real *ranges = (real*)calloc(q.pr,sizeof(*ranges));
  for(i=0;i<q.r;i++){
    ranges[i] = distVec(q,x,i,NNs.mat[IDX(i, K-1, NNs.ld)]);
  }
  
  unint *cnts = (unint*)calloc(q.pr,sizeof(*cnts));
  bruteRangeCount(x,q,ranges,cnts);
  
  nc=0;
  for(i=0;i<m;i++){
    nc += cnts[i];
  }
  mean = ((double)nc)/((double)m);
  var = 0.0;
  for(i=0;i<m;i++) {
    var += (((double)cnts[i])-mean)*(((double)cnts[i])-mean)/((double)m);
  }
  printf("\tavg actual rank of 32nd NN returned by the RBC = %6.4f; std dev = %6.4f \n\n", mean, sqrt(var));
  printf("(brute k-nn took %6.4f) \n", timeDiff(tvB, tvE));

  if(outFile){
    fprintf( fp, "%6.5f %6.5f \n", mean, sqrt(var) );
    fclose(fp);
  }

  free(cnts);
  free(ol);
  free(NNsB.mat);
  free(distsBrute.mat);
}
