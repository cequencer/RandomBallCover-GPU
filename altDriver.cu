#include "hip/hip_runtime.h"
/* This file is part of the Random Ball Cover (RBC) library.
 * (C) Copyright 2010, Lawrence Cayton [lcayton@tuebingen.mpg.de]
 */

#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<sys/time.h>
#include<math.h>
#include "defs.h"
#include "utils.h"
#include "utilsGPU.h"
#include "rbc.h"
#include "brute.h"
#include "sKernel.h"

#include "kernels.h" //delete

void parseInput(int,char**);
void readData(char*,matrix);
void readDataText(char*,matrix);
void evalNNerror(matrix, matrix, unint*);
void evalKNNerror(matrix,unint*,vorStruct);
void writeNeighbs(char*,char*,intMatrix,matrix);

char *dataFileX, *dataFileQ, *dataFileXtxt, *dataFileQtxt, *outFile, *outFiletxt, *nnFile;
char dataFormat = IS_REAL;
char runBrute=0, runEval=0;
unint n=0, m=0, d=0, numReps=0, deviceNum=0;

int main(int argc, char**argv){
  matrix q;
  struct timeval tvB,tvE;
  hipError_t cE;
  vorStruct vorS;

  printf("*****************\n");
  printf("RANDOM BALL COVER\n");
  printf("*****************\n");
  
  parseInput(argc,argv);
  
  gettimeofday( &tvB, NULL );
  /* printf("Using GPU #%d\n",deviceNum);  */
  /* if(hipSetDevice(deviceNum) != hipSuccess){  */
  /*   printf("Unable to select device %d.. exiting. \n",deviceNum);  */
  /*   exit(1);  */
  /* }  */
  
  size_t memFree, memTot;
  hipMemGetInfo(&memFree, &memTot);
  printf("GPU memory free = %lu/%lu (MB) \n",(unsigned long)memFree/(1024*1024),(unsigned long)memTot/(1024*1024));
  gettimeofday( &tvE, NULL );
  printf(" init time: %6.2f \n", timeDiff( tvB, tvE ) );
  
  //Setup matrices
  initMat( &q, m, d );
  q.mat = (real*)calloc( sizeOfMat(q), sizeof(*(q.mat)) );
  
  readData( dataFileQ, q );

  unint *nrs = (unint*)calloc( PAD(m), sizeof(*nrs));
  
  //Try the alternative method out
  hdMatrix hdx;
  hdx.fp = fopen(dataFileX, "rb");
  if( !hdx.fp ){
    fprintf(stderr,"error opening input file\n");
    exit(1);
  }
  hdx.r = n;
  hdx.c = d;
  hdx.format = dataFormat;

  
  printf("[alt]building the rbc..\n");
  gettimeofday( &tvB, NULL );
  //unint ol = (unint)(((double)numReps)*numReps/((double)n));
  buildBigOneShot( hdx, &vorS, numReps, numReps);
  //  buildVorBig( hdx, &vorS, numReps, ol );
  gettimeofday( &tvE, NULL );
  printf( "\t.. build time = %6.4f \n", timeDiff(tvB,tvE) );

  gettimeofday( &tvB, NULL );
  bruteSearch( vorS.r, q,  nrs );
  gettimeofday( &tvE, NULL );
  printf( "\t.. query time for krbc = %6.4f \n", timeDiff(tvB,tvE) );
  
  //EVAL PHASE
  cE = hipGetLastError();
  if( cE != hipSuccess ){
    printf("Execution failed; error type: %s \n", hipGetErrorString(cE) );
  }
  
  if( runEval )
    evalKNNerror(q,nrs,vorS);
  
  destroyVor( &vorS );
  fclose( hdx.fp );

  hipDeviceReset();

  free( nrs );
  free( q.mat );
}


void parseInput(int argc, char **argv){
  int i=1;
  if(argc <= 1){
    printf("\nusage: \n  testRBC -x datafileX -q datafileQ  -n numPts (DB) -m numQueries -d dim -r numReps [-o outFile] [-g GPU num] [-b] [-e] [-c]\n\n");
    printf("\tdatafileX    = binary file containing the database\n");
    printf("\tdatafileQ    = binary file containing the queries\n");
    printf("\tnumPts       = size of database\n");
    printf("\tnumQueries   = number of queries\n");
    printf("\tdim          = dimensionality\n");
    printf("\tnumReps      = number of representatives\n");
    printf("\toutFile      = binary output file (optional)\n");
    printf("\tGPU num      = ID # of the GPU to use (optional) for multi-GPU machines\n");
    printf("\n\tuse -b to run brute force in addition the RBC\n");
    printf("\tuse -e to run the evaluation routine (implicitly runs brute force)\n");
    printf("\tuse -c if data is stored as chars (otherwise assumed to be reals\n");
    printf("\n\n\tTo input/output data in text format (instead of bin), use the \n\t-X and -Q and -O switches in place of -x and -q and -o (respectively).\n");
    printf("\n\n");
    exit(0);
  }
  
  while(i<argc){
    if(!strcmp(argv[i], "-x"))
      dataFileX = argv[++i];
    else if(!strcmp(argv[i], "-q"))
      dataFileQ = argv[++i];
    else if(!strcmp(argv[i], "-X"))
      dataFileX = argv[++i];
    else if(!strcmp(argv[i], "-Q"))
      dataFileQ = argv[++i];
    else if(!strcmp(argv[i], "-t"))
      nnFile = argv[++i];
    else if(!strcmp(argv[i], "-n"))
      n = atoi(argv[++i]);
    else if(!strcmp(argv[i], "-m"))
      m = atoi(argv[++i]);
    else if(!strcmp(argv[i], "-d"))
      d = atoi(argv[++i]);
    else if(!strcmp(argv[i], "-r"))
      numReps = atoi(argv[++i]);
    else if(!strcmp(argv[i], "-o"))
      outFile = argv[++i];
    else if(!strcmp(argv[i], "-O"))
      outFiletxt = argv[++i];
    else if(!strcmp(argv[i], "-g"))
      deviceNum = atoi(argv[++i]);
    else if(!strcmp(argv[i], "-b"))
      runBrute=1;
    else if(!strcmp(argv[i], "-e"))
      runEval=1;
    else if(!strcmp(argv[i], "-c"))
      dataFormat = IS_CHAR;
    else{
      fprintf(stderr,"%s : unrecognized option.. exiting\n",argv[i]);
      exit(1);
    }
    i++;
  }

  if( !n || !m || !d || !numReps  ){
    fprintf(stderr,"more arguments needed.. exiting\n");
    exit(1);
  }
  if( (!dataFileX && !dataFileXtxt) || (!dataFileQ && !dataFileQtxt) ){
    fprintf(stderr,"more arguments needed.. exiting\n");
    exit(1);
  }
  if( (dataFileX && dataFileXtxt) || (dataFileQ && dataFileQtxt) ){
    fprintf(stderr,"you can only give one database file and one query file.. exiting\n");
    exit(1); 
  }
  if(numReps>n){
    fprintf(stderr,"can't have more representatives than points.. exiting\n");
    exit(1);
  }
}


void readData(char *dataFile, matrix x){
  unint i, j;
  FILE *fp;
  unint numRead;

  fp = fopen(dataFile,"rb");
  if(fp==NULL){
    fprintf(stderr,"error opening file.. exiting\n");
    exit(1);
  }
    
  if(dataFormat == IS_REAL){
    for( i=0; i<x.r; i++ ){ //can't load everything in one fread
      //because matrix is padded.
      numRead = fread( &x.mat[IDX( i, 0, x.ld )], sizeof(real), x.c, fp );
      if(numRead != x.c){
	fprintf(stderr,"error reading file.. exiting \n");
	exit(1);
      }
    }
  }
  else{
    char *t = (char*)calloc( x.c, sizeof(*t) );
    for( i=0; i<x.r; i++ ){ //can't load everything in one fread
      //because matrix is padded.
      numRead = fread( t, sizeof(char), x.c, fp );
      if(numRead != x.c){
	fprintf(stderr,"error reading file.. exiting \n");
	exit(1);
      }
      for( j=0; j<x.c; j++ )
	x.mat[IDX( i, j, x.ld )] = (real)t[j];
    }
    free( t );
  }
  fclose(fp);
}


void readDataText(char *dataFile, matrix x){
  FILE *fp;
  real t;
  int i,j;

  fp = fopen(dataFile,"r");
  if(fp==NULL){
    fprintf(stderr,"error opening file.. exiting\n");
    exit(1);
  }
    
  for(i=0; i<x.r; i++){
    for(j=0; j<x.c; j++){
      if(fscanf(fp,"%f ", &t)==EOF){
	fprintf(stderr,"error reading file.. exiting \n");
	exit(1);
      }
      x.mat[IDX( i, j, x.ld )]=(real)t;
    }
  }
  fclose(fp);
}


//find the error rate of a set of NNs, then print it.
void evalNNerror(matrix x, matrix q, unint *NNs){
  struct timeval tvB, tvE;
  unint i;

  printf("\nComputing error rates (this might take a while)\n");
  real *ranges = (real*)calloc(q.pr,sizeof(*ranges));
  for(i=0;i<q.r;i++){
    if(NNs[i]>n) printf("error");
    ranges[i] = distVec(q,x,i,NNs[i]) - 10e-6;
  }

  unint *cnts = (unint*)calloc(q.pr,sizeof(*cnts));
  gettimeofday(&tvB,NULL);
  bruteRangeCount(x,q,ranges,cnts);
  gettimeofday(&tvE,NULL);
  
  long int nc=0;
  for(i=0;i<m;i++){
    nc += cnts[i];
  }
  double mean = ((double)nc)/((double)m);
  double var = 0.0;
  for(i=0;i<m;i++) {
    var += (((double)cnts[i])-mean)*(((double)cnts[i])-mean)/((double)m);
  }
  printf("\tavg rank = %6.4f; std dev = %6.4f \n\n", mean, sqrt(var));
  printf("(range count took %6.4f) \n", timeDiff(tvB, tvE));
  
  if(outFiletxt){
    FILE* fp = fopen(outFiletxt, "a");
    fprintf( fp, "%d %6.5f %6.5f \n", numReps, mean, sqrt(var) );
    fclose(fp);
  }

  free(ranges);
  free(cnts);
}


//evals the error rate of k-nns
void evalKNNerror(matrix q, unint *NNs, vorStruct vorS){
  unint i,j,k,l;

  unint m = q.r;
  printf("\nComputing error rates (this might take a while)\n");
  
  unint *trueNNs = (unint*)calloc( 32*m, sizeof(*trueNNs) );
  FILE *fp = fopen( nnFile, "r" );
  if( 32*m != fread( trueNNs, sizeof(unint), 32*m, fp ) ){
    printf("error reading NN file \n");
    exit(1);
  }
  fclose( fp );

  unint s=vorS.r.r;
  unint *txmap = (unint*)calloc( s, sizeof(*txmap) );

  unint **patk = (unint**)calloc( q.r, sizeof(unint*) );
  for( i=0; i<q.r; i++ )
    patk[i] = (unint*)calloc( KMAX, sizeof(unint*) );
  unint *total = (unint*)calloc( q.r, sizeof(unint) );

  fp = fopen( vorS.filename, "rb" );
  if(!fp){
    printf("error opening vorS file %s\n",vorS.filename);
    exit(1);
  }

  for( i=0; i<q.r; i++ ){
    unint ri = NNs[i];
    total[i] = vorS.groupCount[ri];

    if( fseek( fp, ri*s*sizeof(unint), SEEK_SET ) ){
      fprintf(stderr,"problem with fseek \n");
      exit(1);
    }
      
    if( total[i] != fread( txmap, sizeof(*txmap), total[i], fp ) ){
      fprintf(stderr,"problem reading xmap\n"); 
      exit(1); 
    }

    for( j=0; j<total[i]; j++ )
      for( k=1; k<= KMAX; k++ )
	for( l=0; l<k; l++ )
	  patk[i][k-1] += ( txmap[j] == trueNNs[IDX( i, l, 32 )] );
  }

  fclose(fp);

  long unsigned int *a_patk = (long unsigned int*)calloc(KMAX, sizeof(*a_patk));
  long unsigned int a_total = 0;
  for( i=0; i<q.r; i++ ){
    a_total += total[i];
    for( j=0; j<KMAX; j++ )
      a_patk[j] += patk[i][j];
  }
  
  double *mu = (double*)calloc(KMAX, sizeof(*mu));
  double *sig2 = (double*)calloc(KMAX, sizeof(*sig2));
  
  for( i=0; i<KMAX; i++ )
    mu[i] = ((double)a_patk[i])/((double)q.r);
  for( i=0; i<q.r; i++ ){
    for( j=0; j<KMAX; j++ )
      sig2[j] += ((double)patk[i][j] - mu[j])*((double)patk[i][j] - mu[j])/((double)q.r);
  }
  
  printf("avg patk: ");
  for(i=0; i<KMAX; i++)
    printf("%d: %6.3f ", i, mu[i]);
  printf("\n");
  
  printf("var patk: ");
  for(i=0; i<KMAX; i++)
    printf("%d: %6.3f ", i, sqrt(sig2[i]));
  printf("\n");
  
  double meanT = ((double)a_total)/((double)m);
  double varT = 0.0;
  for(i=0;i<m;i++) {
    varT += (((double)total[i])-meanT)*(((double)total[i])-meanT)/((double)m);
  }
  printf("\tnum dists = %6.4f; std dev = %6.4f \n", meanT, sqrt(varT));
  
  if(outFiletxt){
    FILE* fp = fopen(outFiletxt, "a");
    fprintf( fp, "%d %6.5f %6.5f %6.5f ", numReps, meanT, sqrt(varT), 0.0 );
    for(i=0; i<KMAX; i++)
      fprintf(fp,"%6.5f %6.5f ", mu[i], sqrt(sig2[i]));
    fprintf(fp,"\n");
   
    fclose(fp);
  }

  free(mu);
  free(sig2);
  free(a_patk);
  free(total);
  for( i=0; i<q.r; i++ )
    free(patk[i]);
  free(patk);
  free(txmap);
}


void writeNeighbs(char *file, char *filetxt, intMatrix NNs, matrix dNNs){
  unint i,j;
  
  if( filetxt ) { //write text

    FILE *fp = fopen(filetxt,"w");
    if( !fp ){
      fprintf(stderr, "can't open output file\n");
      return;
    }
    
    for( i=0; i<m; i++ ){
      for( j=0; j<KMAX; j++ )
	fprintf( fp, "%u ", NNs.mat[IDX( i, j, NNs.ld )] );
      fprintf(fp, "\n");
    }
    
    for( i=0; i<m; i++ ){
      for( j=0; j<KMAX; j++ )
	fprintf( fp, "%f ", dNNs.mat[IDX( i, j, dNNs.ld )]); 
      fprintf(fp, "\n");
    }
    fclose(fp);
    
  }

  if( file ){ //write binary

    FILE *fp = fopen(file,"wb");
    if( !fp ){
      fprintf(stderr, "can't open output file\n");
      return;
    }
    
    for( i=0; i<m; i++ )
      fwrite( &NNs.mat[IDX( i, 0, NNs.ld )], sizeof(*NNs.mat), KMAX, fp );
    for( i=0; i<m; i++ )
      fwrite( &dNNs.mat[IDX( i, 0, dNNs.ld )], sizeof(*dNNs.mat), KMAX, fp );
    
    fclose(fp);
  }
}
