#include "hip/hip_runtime.h"
/* This file contains functions that were in use at one point but 
   are not currently used.  As far as I know, everything here is
   debugged, and so can be plugged in reasonably safely.
*/



/* This is the more complex version of computeReps.  It can be used even if X doesn't
fit into the device memory.  It does not work in emulation mode, because it checks to see
how much mem is available on the device.  Thus for debugging purposes we currently 
use a simpler version of computeReps. */

//Assumes that dr is a matrix already on the device
void computeReps(matrix x, matrix dr, int *repIDs, real *distToReps){
  int memPerX, segSize; //seg==segment
  int index, tempSize; //temp variables used in the loop
  int i;
  matrix dx;
  real *dMins;
  int *dMinIDs;
  memPlan mp;

  int n = x.r; //For convenience
  
  // Items that need to go on device: x, repIDs, distToReps.  The "+1" is for the
  // distance from each point to its nearest rep (distToReps) and the int is for
  // the ID (repIDs).
  memPerX = (x.pc+1)*sizeof(real)+sizeof(int);
  mp = createMemPlan(x.r,memPerX);
  
  for(i=0;i<mp.numComputeSegs;i++){
    if(i==mp.numComputeSegs-1)
      segSize = mp.lastSegSize;
    else
      segSize = mp.normSegSize;

    //Allocate & copy over data
    index = IDX(mp.normSegSize*i,0,x.ld);
    tempSize = segSize*x.pc*sizeof(*(dx.mat));

    hipMalloc((void**)&(dx.mat),tempSize);
    hipMemcpy(dx.mat,&(x.mat[index]),tempSize,hipMemcpyHostToDevice);
    dx.r=segSize; dx.c=x.c; dx.pr=dx.r; dx.pc=x.pc; dx.ld=x.ld;

    //Allocate matrices to temporarily store mins and IDs (NOTE:MOVE OUT OF LOOP FOR EFFICIENCY)
    hipMalloc((void**)&(dMins), PAD(MIN(segSize,n))*sizeof(*dMins));
    hipMalloc((void**)&(dMinIDs), PAD(MIN(segSize,n))*sizeof(*dMinIDs));
    nnWrap(dx,dr,dMins,dMinIDs);

    hipMemcpy(&distToReps[i*segSize],dMins,MIN(segSize,n)*sizeof(*dMins),hipMemcpyDeviceToHost);
    hipMemcpy(&repIDs[i*segSize],dMinIDs,MIN(segSize,n)*sizeof(*dMinIDs),hipMemcpyDeviceToHost);
    
    hipFree(dMins);
    hipFree(dMinIDs);
    hipFree(dx.mat);
  }
}


__global__ void getMinsKernel(matrix,real*,int*);

// Returns the min of each row of D.  dMins and dMinIDs 
// are assumed to be (at least) of size D.r.
__global__ void getMinsKernel(const matrix D, real *dMins, int *dMinIDs){
  int row, locRow, colOff, i, curCol;
  real temp;

  row = blockIdx.y*BLOCK_SIZE+threadIdx.y;
  locRow = threadIdx.y;
  
  colOff = threadIdx.x; //column offset of this thread
 
  __shared__ float mins[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ int pos[BLOCK_SIZE][BLOCK_SIZE];
  

  // This loop finds the minimum of cols 
  // [colOff, BLOCK_SIZE+colOff, 2*BLOCK_SIZE+colOff,...]
  // and stores it in mins[locRow][colOff].
  mins[locRow][colOff]=MAX_REAL;
  pos[locRow][colOff]=-1;
  for (i=0;i<D.pc/BLOCK_SIZE;i++){
    curCol = i*BLOCK_SIZE+colOff;
    if(curCol < D.c){ //ignore padding
      temp = D.mat[IDX(row,curCol,D.ld)];
      if(temp<mins[locRow][colOff]){
	mins[locRow][colOff]=temp;
	pos[locRow][colOff]=curCol;
      }
    }
  }
  __syncthreads();
    
  //Now find the min of cols [0, ... , BLOCK_SIZE]
  for (i=BLOCK_SIZE/2; i>0;i/=2){
    if(colOff<i){
      //compare (col) to (col+i)
      if(mins[locRow][colOff]>mins[locRow][colOff+i]){
	mins[locRow][colOff]=mins[locRow][colOff+i];
	pos[locRow][colOff]=pos[locRow][colOff+i];
      }
    }
    __syncthreads();
  }
  
  //arbitrarily use the first thread (along x) to set memory
  if(threadIdx.x==0){  
    dMins[row] = mins[locRow][0];
    dMinIDs[row] = pos[locRow][0];
  }
}


// Returns the min of each row of D.  dMins and dMinIDs 
// are assumed to be (at least) of size D.r.
__global__ void getKMinsKernel(matrix D, matrix dMins, intMatrix NNs, int k){
  int row, locRow, colOff, i, curCol,j;
  real temp;

  row = blockIdx.y*BLOCK_SIZE+threadIdx.y;
  locRow = threadIdx.y;

  //printf("row=%d D.r =%d \n",row,D.r);
  /* if(row>=D.r) */
  /*   return; */

  colOff = threadIdx.x; //column offset of this thread
 
  __shared__ float mins[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ int pos[BLOCK_SIZE][BLOCK_SIZE];
  
  for(i=0;i<k;i++){
    // This loop finds the minimum of cols 
    // [colOff, BLOCK_SIZE+colOff, 2*BLOCK_SIZE+colOff,...]
    // and stores it in mins[locRow][colOff].
    mins[locRow][colOff]=MAX_REAL;
    pos[locRow][colOff]=-1;
    for (j=0;j<D.pc/BLOCK_SIZE;j++){
      curCol = j*BLOCK_SIZE+colOff;
      if(curCol < D.c){ //ignore padding
	temp = D.mat[IDX(row,curCol,D.ld)];
	if(temp<mins[locRow][colOff]){
	  mins[locRow][colOff]=temp;
	  pos[locRow][colOff]=curCol;
	}
      }
    }
    __syncthreads();


    //Now find the min of cols [0, ... , BLOCK_SIZE]
    for (j=BLOCK_SIZE/2; j>0; j/=2){
      if(colOff<j){	
	//compare (col) to (col+j)
	if(mins[locRow][colOff]>mins[locRow][colOff+j]){
	  mins[locRow][colOff]=mins[locRow][colOff+j];
	  pos[locRow][colOff]=pos[locRow][colOff+j];
	}
      }
       __syncthreads();
    }
    
  //arbitrarily use the first thread (along x) to set memory
    if(threadIdx.x==0 && row<D.r){  
      dMins.mat[IDX(row,i,dMins.ld)] = mins[locRow][0];
      NNs.mat[IDX(row,i,NNs.ld)] = pos[locRow][0];
      D.mat[IDX(row,pos[locRow][0],D.ld)]=MAX_REAL;
      
    }
    __syncthreads();
  }
}

size_t countCompute(int*,int*,charMatrix);

//This is used for debugging/research.
size_t countCompute(int *groupCountQ, int *groupCountX, charMatrix cM){
  int i,j;
  size_t ans=0;
  size_t avgBlocks=0;
  size_t maxBlocks=0;
  size_t maxBlocksInd;
  size_t maxTemp;
  size_t avgBlockQ=0;
  size_t avgBlockX=0;
  size_t maxBlockX=0;
  size_t maxBlockQ=0;


  for(i=0;i<cM.c;i++){
    maxTemp=0;
    for(j=0;j<cM.r;j++){
      //printf("%d ",cM.mat[IDX(i,j,cM.ld)]*PAD(groupCountQ[i])*PAD(groupCountX[j]));
      ans+=cM.mat[IDX(i,j,cM.ld)]*(groupCountQ[i])*(groupCountX[j]);
      avgBlocks+=cM.mat[IDX(i,j,cM.ld)];
      maxTemp+=cM.mat[IDX(i,j,cM.ld)]*PAD(groupCountX[j]);
    }
    //    printf("\n");
    if(maxBlocks < maxTemp){
      maxBlocks=maxTemp;
      maxBlocksInd=PAD(groupCountQ[i]);
    }
    //maxBlocks=MAX(maxTemp,maxBlocks);
  }
  
  for(i=0;i<cM.c;i++){
    avgBlockQ+=groupCountQ[i];
    avgBlockX+=groupCountX[i];
    maxBlockQ=MAX(maxBlockQ,groupCountQ[i]);
    maxBlockX=MAX(maxBlockX,groupCountX[i]);
  }
  
  printf("most amt of work for a query: %zu (%zu) ; avg = %6.4f \n",maxBlocks,maxBlocksInd,((double)ans)/((double)cM.c));
  printf("avg blocks/query block = %6.4f ; \n",((double)avgBlocks)/((double)cM.c));
  printf("avg blockQ = %6.4f; max = %zu \n",((double)avgBlockQ)/((double)cM.c),maxBlockQ);
  printf("avg blockX = %6.4f; max = %zu \n",((double)avgBlockX)/((double)cM.c),maxBlockX);
  
  return ans;
}


void kMinsWrap(matrix dD, matrix dMins, intMatrix dNNs){
  dim3 block(BLOCK_SIZE,BLOCK_SIZE);
  dim3 grid(1,dD.pr/BLOCK_SIZE);
  
  kMinsKernel<<<grid,block>>>(dD,dMins,dNNs);
  hipDeviceSynchronize();
}


__global__ void kMinsKernel(matrix,matrix,intMatrix);

__global__ void kMinsKernel(matrix D, matrix dMins, intMatrix NNs){
  
  int row = blockIdx.y*BLOCK_SIZE + threadIdx.y;
  int ro = threadIdx.y; //row offset
  int co = threadIdx.x; //col offset

  __shared__ real smin[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ int smp[BLOCK_SIZE][BLOCK_SIZE];

  real min, t;
  int mp; //min position
  int i,j,c;
  
   
  for(i=0 ; i<NNs.c ; i++){
    
    min = MAX_REAL;
    for(j=0 ; j<D.pc/BLOCK_SIZE ; j++){
      c = j*BLOCK_SIZE;
      if( c+co < D.c ){
	t = D.mat[ IDX( row, c+co, D.ld ) ];
	
	if( t < min ){
	  min = t;
	  mp = c+co;
	}
      }
    }
    
    smin[ro][co] = min;
    smp[ro][co] = mp;
    __syncthreads();


    for(j=BLOCK_SIZE/2 ; j>0 ; j/=2){
      if( co < j ){
	if( smin[ro][co+j] < smin[ro][co] ){
	  smin[ro][co] = smin[ro][co+j];
	  smp[ro][co] = smp[ro][co+j];
	}
      }
      __syncthreads();
    }

    if(co==0){
      NNs.mat[ IDX( row, i, NNs.ld ) ] = smp[ro][0];
      dMins.mat[ IDX( row, i, dMins.ld ) ] = smin[ro][0];

      D.mat[ IDX( row, smp[ro][0], D.ld ) ] = MAX_REAL;
    }

    __syncthreads();
  }
}

void brute2Step(matrix,matrix,intMatrix);


void brute2Step(matrix x, matrix q, intMatrix NNs){

  matrix dx, dq, dD, dMins;
  intMatrix dNNs;
  real *ranges, *dranges;
  
  copyAndMove(&dx, &x);
  copyAndMove(&dq, &q);
  copyAndMoveI(&dNNs, &NNs); //NNs.mat is garbage, but no matter.
  
  dD.r=q.r; dD.pr=q.pr; dD.c=x.r; dD.pc=x.pr; dD.ld=dD.pc;
  hipMalloc( (void**)&dD.mat, dD.pr*dD.pc*sizeof(*dD.mat) );
  
  dMins.r=NNs.r; dMins.pr=NNs.pr; dMins.c=NNs.c; dMins.pc=NNs.pc; dMins.ld=NNs.ld;
  hipMalloc( (void**)&dMins.mat, dMins.pr*dMins.pc*sizeof(*dMins.mat) );
  
  ranges = (real*)calloc( q.pr, sizeof(*ranges) );
  hipMalloc( (void**)&dranges, q.pr*sizeof(*dranges) );

  dist1Wrap(dq,dx,dD);

  kMinsWrap(dD, dMins, dNNs);
  hipMemcpy(NNs.mat, dNNs.mat, NNs.pr*NNs.pc*sizeof(*NNs.mat), hipMemcpyDeviceToHost);
 
  free(ranges);
  hipFree(dranges);
  hipFree(dx.mat);
  hipFree(dq.mat);
  hipFree(dD.mat);
  hipFree(dNNs.mat);
  hipFree(dMins.mat);

}


memPlan createMemPlan(int,int);

memPlan createMemPlan(unint nPts, unint memPerPt){
  memPlan mp;
  unsigned int memFree, memTot;
  unint ptsAtOnce;

  hipMemGetInfo(&memFree, &memTot);
  memFree = (unint)(((float)memFree)*MEM_USABLE);
  printf("memfree = %d \n",memFree);
  ptsAtOnce = DPAD(memFree/memPerPt); //max number of pts that can be processed at once
  printf("ptsAtOnce = %d \n",ptsAtOnce);
  mp.numComputeSegs = nPts/ptsAtOnce + ((nPts%ptsAtOnce==0) ? 0 : 1);
  mp.normSegSize=PAD(nPts/mp.numComputeSegs); 
  mp.lastSegSize=PAD(nPts) - mp.normSegSize*(mp.numComputeSegs-1);
  //Note that lastSegSize is automatically padded if nPts is.
  return mp;
}

typedef struct {
  unint numComputeSegs;
  unint normSegSize;//The number of points handled in one computation,
                     //though there will always be one leftover segment
                     //with (possibly) a different number of points.
  unint lastSegSize;//.. and this is it.
} memPlan;


void blockIntersection(charMatrix,matrix,real*,real*);

void blockIntersection(charMatrix cM, matrix dr, real *radiiX, real *radiiQ){
  matrix dD;
  real *dradiiX, *dradiiQ;
  unint pnR = dr.pr;
  charMatrix dcM;
  
  dD.r=dD.c=dr.r; dD.pr=dD.pc=dD.ld=dr.pr;
  dcM.r=cM.r; dcM.c=cM.c; dcM.pr=cM.pr; dcM.pc=cM.pc; dcM.ld=cM.ld;
  
  checkErr( hipMalloc((void**)&dD.mat, pnR*pnR*sizeof(*dD.mat)) );
  checkErr( hipMalloc((void**)&dradiiX, pnR*sizeof(*dradiiX)) );
  checkErr( hipMalloc((void**)&dradiiQ, pnR*sizeof(*dradiiQ)) );
  checkErr( hipMalloc((void**)&dcM.mat, dcM.pr*dcM.pc*sizeof(*dcM.mat)) );
  
  // Copying over the radii. Note that everything after the first dr.r places 
  // on the device variables is undefined.
  hipMemcpy(dradiiX,radiiX,dr.r*sizeof(*dradiiX),hipMemcpyHostToDevice);
  hipMemcpy(dradiiQ,radiiQ,dr.r*sizeof(*dradiiQ),hipMemcpyHostToDevice);
  
  dist1Wrap(dr, dr, dD);
  pruneWrap(dcM, dD, dradiiX, dradiiQ);

  hipMemcpy(cM.mat,dcM.mat,pnR*pnR*sizeof(*dcM.mat),hipMemcpyDeviceToHost);
  
  hipFree(dcM.mat);
  hipFree(dradiiQ);
  hipFree(dradiiX);
  hipFree(dD.mat);
}



void groupPoints(matrix,unint*,unint*,unint);

// This function sorts points by their repID.  It makes two passes through the 
// matrix x; one to count the bucket sizes, the next to place points in the 
// correct bucket.  Note that this function allocates a temporary
// matrix the size of x, then copies the results over to x at the end.  The 
// sort could be done in place, eg by doing numReps passes through x instead of 2.
void groupPoints(matrix x, unint *xID, unint *repIDs, unint numReps){
  matrix y;
  unint n=x.r;
  unint d=x.c;
  unint i;
  unint *gS; //groupSize
  unint *yID;

  yID = (unint*)calloc(n,sizeof(*yID));
  y.mat = (real*)calloc(n*d,sizeof(*y.mat));
  gS = (unint*)calloc(numReps+1,sizeof(*gS));

  y.r=n; y.pr=n; y.c=d; y.pc=d; y.ld=d;

  for(i=0;i<n;i++)
    gS[repIDs[i]+1]++;
  for(i=1;i<numReps;i++)
    gS[i]=gS[i-1]+gS[i];
  
  for(i=0;i<n;i++){
    copyVector(&y.mat[IDX(gS[repIDs[i]],0,y.ld)], &x.mat[IDX(i,0,x.ld)],d);
    yID[gS[repIDs[i]]]=xID[i];
    gS[repIDs[i]]++;
  }
  
  for(i=0;i<n;i++){
    copyVector(&x.mat[IDX(i,0,x.ld)],&y.mat[IDX(i,0,y.ld)],d);
    xID[i]=yID[i];
  }
  
  free(yID);
  free(gS);
  free(y.mat);
}

__global__ void pruneKernel(const matrix,const real*,const real*,charMatrix);


__global__ void pruneKernel(const matrix D, const real *radiiX, const real *radiiQ, charMatrix cM){
  unint offX = threadIdx.x;
  unint offQ = threadIdx.y;

  unint blockX = blockIdx.x * BLOCK_SIZE;
  unint blockQ = blockIdx.y * BLOCK_SIZE;
  
  __shared__ real sD[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ real sRQ[BLOCK_SIZE];
  __shared__ real sRX[BLOCK_SIZE];

  sD[offQ][offX]=D.mat[IDX(blockQ+offQ,blockX+offX,D.ld)];
  
  if(offQ==0)
    sRX[offX]=radiiX[blockX+offX];
  if(offX==0)
    sRQ[offQ]=radiiQ[blockQ+offQ];
  
  __syncthreads();
  
  if(blockQ+offQ < D.r && blockX+offX < D.c){
    cM.mat[IDX(blockQ+offQ,blockX+offX,cM.ld)] = (sD[offQ][offX]-sRX[offX]-2*sRQ[offQ] <= 0) ? 1 : 0;
  }
}

void pruneWrap(charMatrix dcM, matrix dD, real *dradiiX, real *dradiiQ){
  dim3 block(BLOCK_SIZE,BLOCK_SIZE);
  dim3 grid(dD.pr/BLOCK_SIZE,dD.pc/BLOCK_SIZE);
  
  pruneKernel<<<grid,block>>>(dD,dradiiX,dradiiQ,dcM);
  hipDeviceSynchronize();
}

void pruneWrap(charMatrix,matrix,real*,real*);
