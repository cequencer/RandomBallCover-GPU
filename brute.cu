/* This file is part of the Random Ball Cover (RBC) library.
 * (C) Copyright 2010, Lawrence Cayton [lcayton@tuebingen.mpg.de]
 */

#ifndef BRUTE_CU
#define BRUTE_CU

#include "utilsGPU.h"
#include "utils.h"
#include "rbc.h"
#include "defs.h"
#include "kernels.h"
#include "kernelWrap.h"
#include "brute.h"
#include<stdio.h>
#include<hip/hip_runtime.h>

void bruteRangeCount(matrix x, matrix q, real *ranges, unint *cnts){
  matrix dx, dq;
  real *dranges;
  unint *dcnts;
  
  copyAndMove(&dx, &x);
  copyAndMove(&dq, &q);

  checkErr( hipMalloc( (void**)&dranges, q.pr*sizeof(*dranges) ) );
  hipMemcpy( dranges, ranges, q.r*sizeof(*dranges), hipMemcpyHostToDevice );

  checkErr( hipMalloc( (void**)&dcnts, q.pr*sizeof(*dcnts) ) );
  
  rangeCountWrap(dq, dx, dranges, dcnts);
  
  hipMemcpy(cnts, dcnts, q.r*sizeof(*cnts), hipMemcpyDeviceToHost );

  hipFree(dcnts);
  hipFree(dranges);
  hipFree(dx.mat);
  hipFree(dq.mat);
}


void bruteSearch(matrix x, matrix q, unint *NNs){
  real *dMins;
  unint *dMinIDs;
  matrix dx, dq;

  
  dx.r=x.r; dx.pr=x.pr; dx.c=x.c; dx.pc=x.pc; dx.ld=x.ld;
  dq.r=q.r; dq.pr=q.pr; dq.c=q.c; dq.pc=q.pc; dq.ld=q.ld;

  checkErr( hipMalloc((void**)&dMins, q.pr*sizeof(*dMins)) );
  checkErr( hipMalloc((void**)&dMinIDs, q.pr*sizeof(*dMinIDs)) );
  checkErr( hipMalloc((void**)&dx.mat, dx.pr*dx.pc*sizeof(*dx.mat)) );
  checkErr( hipMalloc((void**)&dq.mat, dq.pr*dq.pc*sizeof(*dq.mat)) );

  hipMemcpy(dx.mat,x.mat,x.pr*x.pc*sizeof(*dx.mat),hipMemcpyHostToDevice);
  hipMemcpy(dq.mat,q.mat,q.pr*q.pc*sizeof(*dq.mat),hipMemcpyHostToDevice);
  
  nnWrap(dq,dx,dMins,dMinIDs);

  hipMemcpy(NNs,dMinIDs,dq.r*sizeof(*NNs),hipMemcpyDeviceToHost);
  
  hipFree(dMins);
  hipFree(dMinIDs);
  hipFree(dx.mat);
  hipFree(dq.mat);
}


void bruteCPU(matrix X, matrix Q, unint *NNs){
  real *dtoNNs; 
  real temp;

  unint i, j;

  dtoNNs = (real*)calloc(Q.r,sizeof(*dtoNNs));
  
  for( i=0; i<Q.r; i++ ){
    dtoNNs[i] = MAX_REAL;
    NNs[i] = 0;
    for(j=0; j<X.r; j++ ){
      temp = distVec( Q, X, i, j );
      if( temp < dtoNNs[i]){
	NNs[i] = j;
	dtoNNs[i] = temp;
      }
    }
  }
  
  free(dtoNNs);  
}
#endif
