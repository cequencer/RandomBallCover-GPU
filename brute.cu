/* This file is part of the Random Ball Cover (RBC) library.
 * (C) Copyright 2010, Lawrence Cayton [lcayton@tuebingen.mpg.de]
 */

#ifndef BRUTE_CU
#define BRUTE_CU

#include "utilsGPU.h"
#include "utils.h"
#include "rbc.h"
#include "defs.h"
#include "kernels.h"
#include "kernelWrap.h"
#include "brute.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
//#include<gsl/gsl_sort.h>

void bruteRangeCount(matrix x, matrix q, real *ranges, unint *cnts){
  matrix dx, dq;
  real *dranges;
  unint *dcnts;
  
  copyAndMove(&dx, &x);
  copyAndMove(&dq, &q);

  checkErr( hipMalloc( (void**)&dranges, q.pr*sizeof(*dranges) ) );
  hipMemcpy( dranges, ranges, q.r*sizeof(*dranges), hipMemcpyHostToDevice );

  checkErr( hipMalloc( (void**)&dcnts, q.pr*sizeof(*dcnts) ) );
  
  rangeCountWrap(dq, dx, dranges, dcnts);
  
  hipMemcpy(cnts, dcnts, q.r*sizeof(*cnts), hipMemcpyDeviceToHost );

  hipFree(dcnts);
  hipFree(dranges);
  hipFree(dx.mat);
  hipFree(dq.mat);
}


void bruteSearch(matrix x, matrix q, unint *NNs){
  real *dMins;
  unint *dMinIDs;
  matrix dx, dq;

  copyAndMove( &dx, &x );
  copyAndMove( &dq, &q );

  checkErr( hipMalloc((void**)&dMins, q.pr*sizeof(*dMins)) );
  checkErr( hipMalloc((void**)&dMinIDs, q.pr*sizeof(*dMinIDs)) );
  
  nnWrap( dq, dx, dMins, dMinIDs );

  hipMemcpy( NNs, dMinIDs, dq.r*sizeof(*NNs), hipMemcpyDeviceToHost );
  
  hipFree( dMins );
  hipFree( dMinIDs );
  hipFree( dx.mat );
  hipFree( dq.mat );
}


void bruteK(matrix x, matrix q, intMatrix NNs, matrix NNdists){
  matrix dNNdists;
  intMatrix dMinIDs;
  matrix dx, dq;
  
  copyAndMove( &dx, &x );
  copyAndMove( &dq, &q );
  
  initMat( &dNNdists, q.r, KMAX );
  checkErr( hipMalloc((void**)&dNNdists.mat, sizeOfMatB(dNNdists) ) );

  initIntMat( &dMinIDs, q.r, KMAX );
  checkErr( hipMalloc((void**)&dMinIDs.mat, sizeOfIntMatB(dMinIDs) ) );

  knnWrap( dq, dx, dNNdists, dMinIDs );

  hipMemcpy( NNs.mat, dMinIDs.mat, sizeOfIntMatB(NNs), hipMemcpyDeviceToHost );
  hipMemcpy( NNdists.mat, dNNdists.mat, sizeOfMatB(NNdists), hipMemcpyDeviceToHost );
  
  hipFree( dNNdists.mat );
  hipFree( dMinIDs.mat );
  hipFree( dx.mat );
  hipFree( dq.mat );
}


void bruteCPU(matrix X, matrix Q, unint *NNs){
  real *dtoNNs; 
  real temp;

  unint i, j;

  dtoNNs = (real*)calloc(Q.r,sizeof(*dtoNNs));
  
  for( i=0; i<Q.r; i++ ){
    dtoNNs[i] = MAX_REAL;
    NNs[i] = 0;
    for(j=0; j<X.r; j++ ){
      temp = distVec( Q, X, i, j );
      if( temp < dtoNNs[i]){
	NNs[i] = j;
	dtoNNs[i] = temp;
      }
    }
  }
  
  free(dtoNNs);  
}


//The following method works properly, but requires the GNU scientific
//library.  If you want to use it, uncomment the code, uncomment the include
//above, and adjust the makefile.  It was used for debugging purposes, but
//is not required by anything else.

/* void bruteKCPU(matrix x, matrix q, intMatrix NNs){ */
/*   int i, j; */

/*   float **d; */
/*   d = (float**)calloc(q.pr, sizeof(*d)); */
/*   size_t **t; */
/*   t = (size_t**)calloc(q.pr, sizeof(*t)); */
/*   for( i=0; i<q.pr; i++){ */
/*     d[i] = (float*)calloc(x.pr, sizeof(**d)); */
/*     t[i] = (size_t*)calloc(x.pr, sizeof(**t)); */
/*   } */

/*   for( i=0; i<q.r; i++){ */
/*     for( j=0; j<x.r; j++) */
/*       d[i][j] = distVec( q, x, i, j ); */
/*     gsl_sort_float_index(t[i], d[i], 1, x.r); */
/*     for ( j=0; j<KMAX; j++) */
/*       NNs.mat[IDX( i, j, NNs.ld )] = t[i][j]; */
/*   } */

/*   for( i=0; i<q.pr; i++){ */
/*     free(t[i]); */
/*     free(d[i]); */
/*   } */
/*   free(t); */
/*   free(d); */
/* } */
#endif
