/* This file is part of the Random Ball Cover (RBC) library.
 * (C) Copyright 2010, Lawrence Cayton [lcayton@tuebingen.mpg.de]
 */

#ifndef BRUTE_CU
#define BRUTE_CU

#include "utilsGPU.h"
#include "utils.h"
#include "rbc.h"
#include "defs.h"
#include "kernels.h"
#include "kernelWrap.h"
#include "brute.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<gsl/gsl_sort.h>

void bruteRangeCount(matrix x, matrix q, real *ranges, unint *cnts){
  matrix dx, dq;
  real *dranges;
  unint *dcnts;
  
  copyAndMove(&dx, &x);
  copyAndMove(&dq, &q);

  checkErr( hipMalloc( (void**)&dranges, q.pr*sizeof(*dranges) ) );
  hipMemcpy( dranges, ranges, q.r*sizeof(*dranges), hipMemcpyHostToDevice );

  checkErr( hipMalloc( (void**)&dcnts, q.pr*sizeof(*dcnts) ) );
  
  rangeCountWrap(dq, dx, dranges, dcnts);
  
  hipMemcpy(cnts, dcnts, q.r*sizeof(*cnts), hipMemcpyDeviceToHost );

  hipFree(dcnts);
  hipFree(dranges);
  hipFree(dx.mat);
  hipFree(dq.mat);
}


void bruteSearch(matrix x, matrix q, unint *NNs){
  real *dMins;
  unint *dMinIDs;
  matrix dx, dq;

  
  dx.r=x.r; dx.pr=x.pr; dx.c=x.c; dx.pc=x.pc; dx.ld=x.ld;
  dq.r=q.r; dq.pr=q.pr; dq.c=q.c; dq.pc=q.pc; dq.ld=q.ld;

  checkErr( hipMalloc((void**)&dMins, q.pr*sizeof(*dMins)) );
  checkErr( hipMalloc((void**)&dMinIDs, q.pr*sizeof(*dMinIDs)) );
  checkErr( hipMalloc((void**)&dx.mat, dx.pr*dx.pc*sizeof(*dx.mat)) );
  checkErr( hipMalloc((void**)&dq.mat, dq.pr*dq.pc*sizeof(*dq.mat)) );

  hipMemcpy(dx.mat,x.mat,x.pr*x.pc*sizeof(*dx.mat),hipMemcpyHostToDevice);
  hipMemcpy(dq.mat,q.mat,q.pr*q.pc*sizeof(*dq.mat),hipMemcpyHostToDevice);
  
  nnWrap(dq,dx,dMins,dMinIDs);

  hipMemcpy(NNs,dMinIDs,dq.r*sizeof(*NNs),hipMemcpyDeviceToHost);
  
  hipFree(dMins);
  hipFree(dMinIDs);
  hipFree(dx.mat);
  hipFree(dq.mat);
}


void bruteK(matrix x, matrix q, intMatrix NNs){
  matrix dMins;
  intMatrix dMinIDs;
  matrix dx, dq;
  
  dx.r=x.r; dx.pr=x.pr; dx.c=x.c; dx.pc=x.pc; dx.ld=x.ld;
  dq.r=q.r; dq.pr=q.pr; dq.c=q.c; dq.pc=q.pc; dq.ld=q.ld;
  dMins.r=q.r; dMins.pr=q.pr; dMins.c=K; dMins.pc=K; dMins.ld=dMins.pc;
  dMinIDs.r=q.r; dMinIDs.pr=q.pr; dMinIDs.c=K; dMinIDs.pc=K; dMinIDs.ld=dMinIDs.pc;

  checkErr( hipMalloc((void**)&dMins.mat, dMins.pc*dMins.pr*sizeof(*dMins.mat)) );
  checkErr( hipMalloc((void**)&dMinIDs.mat, dMinIDs.pc*dMinIDs.pr*sizeof(*dMinIDs.mat)) );
  checkErr( hipMalloc((void**)&dx.mat, dx.pr*dx.pc*sizeof(*dx.mat)) );
  checkErr( hipMalloc((void**)&dq.mat, dq.pr*dq.pc*sizeof(*dq.mat)) );

  hipMemcpy(dx.mat,x.mat,x.pr*x.pc*sizeof(*dx.mat),hipMemcpyHostToDevice);
  hipMemcpy(dq.mat,q.mat,q.pr*q.pc*sizeof(*dq.mat),hipMemcpyHostToDevice);
  
  knnWrap(dq,dx,dMins,dMinIDs);

  hipMemcpy(NNs.mat,dMinIDs.mat,NNs.pr*NNs.pc*sizeof(*NNs.mat),hipMemcpyDeviceToHost);
  
  hipFree(dMins.mat);
  hipFree(dMinIDs.mat);
  hipFree(dx.mat);
  hipFree(dq.mat);
}


void bruteCPU(matrix X, matrix Q, unint *NNs){
  real *dtoNNs; 
  real temp;

  unint i, j;

  dtoNNs = (real*)calloc(Q.r,sizeof(*dtoNNs));
  
  for( i=0; i<Q.r; i++ ){
    dtoNNs[i] = MAX_REAL;
    NNs[i] = 0;
    for(j=0; j<X.r; j++ ){
      temp = distVec( Q, X, i, j );
      if( temp < dtoNNs[i]){
	NNs[i] = j;
	dtoNNs[i] = temp;
      }
    }
  }
  
  free(dtoNNs);  
}


void bruteKCPU(matrix x, matrix q, intMatrix NNs){
  int i, j;

  float **d;
  d = (float**)calloc(q.pr, sizeof(*d));
  size_t **t;
  t = (size_t**)calloc(q.pr, sizeof(*t));
  for( i=0; i<q.pr; i++){
    d[i] = (float*)calloc(x.pr, sizeof(**d));
    t[i] = (size_t*)calloc(x.pr, sizeof(**t));
  }

  //#pragma omp parallel for private(j)
  for( i=0; i<q.r; i++){
    for( j=0; j<x.r; j++)
      d[i][j] = distVec( q, x, i, j );
    gsl_sort_float_index(t[i], d[i], 1, x.r);
    for ( j=0; j<K; j++)
      NNs.mat[IDX( i, j, NNs.ld )] = t[i][j];
  }

  for( i=0; i<q.pr; i++){
    free(t[i]);
    free(d[i]);
  }
  free(t);
  free(d);
}
#endif
