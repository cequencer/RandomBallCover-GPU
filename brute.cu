#ifndef BRUTE_CU
#define BRUTE_CU

#include "utilsGPU.h"
#include "utils.h"
#include "rbc.h"
#include "defs.h"
#include "kernels.h"
#include "kernelWrap.h"
#include "brute.h"
#include<stdio.h>
#include<hip/hip_runtime.h>

void bruteRangeCount(matrix x, matrix q, real *ranges, int *cnts){
  matrix dx, dq;
  real* dranges;
  int *dcnts;
  
  copyAndMove(&dx, &x);
  copyAndMove(&dq, &q);

  hipMalloc( (void**)&dranges, q.pr*sizeof(*dranges) );
  hipMemcpy( dranges, ranges, q.r*sizeof(*dranges), hipMemcpyHostToDevice );

  hipMalloc( (void**)&dcnts, q.pr*sizeof(*dcnts) );
  
  rangeCountWrap(dq, dx, dranges, dcnts);
  
  hipMemcpy(cnts, dcnts, q.r*sizeof(*cnts), hipMemcpyDeviceToHost );

  hipFree(dcnts);
  hipFree(dranges);
  hipFree(dx.mat);
  hipFree(dq.mat);
}


void bruteSearch(matrix x, matrix q, int *NNs){
  real *dMins;
  int *dMinIDs;
  matrix dx, dq;

  
  dx.r=x.r; dx.pr=x.pr; dx.c=x.c; dx.pc=x.pc; dx.ld=x.ld;
  dq.r=q.r; dq.pr=q.pr; dq.c=q.c; dq.pc=q.pc; dq.ld=q.ld;

  hipMalloc((void**)&dMins, q.pr*sizeof(*dMins));
  hipMalloc((void**)&dMinIDs, q.pr*sizeof(*dMinIDs));
  hipMalloc((void**)&dx.mat, dx.pr*dx.pc*sizeof(*dx.mat));
  hipMalloc((void**)&dq.mat, dq.pr*dq.pc*sizeof(*dq.mat));

  hipMemcpy(dx.mat,x.mat,x.pr*x.pc*sizeof(*dx.mat),hipMemcpyHostToDevice);
  hipMemcpy(dq.mat,q.mat,q.pr*q.pc*sizeof(*dq.mat),hipMemcpyHostToDevice);
  
  nnWrap(dq,dx,dMins,dMinIDs);

  hipMemcpy(NNs,dMinIDs,dq.r*sizeof(*NNs),hipMemcpyDeviceToHost);
  
  hipFree(dMins);
  hipFree(dMinIDs);
  hipFree(dx.mat);
  hipFree(dq.mat);

}

void bruteCPU(matrix X, matrix Q, int *NNs){
  real *dtoNNs; 
  real temp;

  int i, j;

  dtoNNs = (real*)calloc(Q.r,sizeof(*dtoNNs));
  
  for( i=0; i<Q.r; i++ ){
    dtoNNs[i] = MAX_REAL;
    NNs[i] = 0;
    for(j=0; j<X.r; j++ ){
      temp = distL1( Q, X, i, j );
      if( temp < dtoNNs[i]){
	NNs[i] = j;
	dtoNNs[i] = temp;
      }
    }
  }
  
  free(dtoNNs);  
}
#endif
