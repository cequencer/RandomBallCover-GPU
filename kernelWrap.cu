#include "hip/hip_runtime.h"
#ifndef KERNELWRAP_CU
#define KERNELWRAP_CU

#include<hip/hip_runtime.h>
#include<stdio.h>
#include "kernels.h"
#include "defs.h"

void dist1Wrap(matrix dq, matrix dx, matrix dD){
  dim3 block(BLOCK_SIZE,BLOCK_SIZE);
  dim3 grid;
  
  int todoX, todoY, numDoneX, numDoneY;

  numDoneX = 0;
  while ( numDoneX < dx.pr ){
    todoX = min( dx.pr - numDoneX, MAX_BS*BLOCK_SIZE );
    grid.x = todoX/BLOCK_SIZE;
    numDoneY = 0;
    while( numDoneY < dq.pr ){
      todoY = min( dq.pr - numDoneY, MAX_BS*BLOCK_SIZE );
      grid.y = todoY/BLOCK_SIZE;
      dist1Kernel<<<grid,block>>>(dq, numDoneY, dx, numDoneX, dD);
      numDoneY += todoY;
    }
    numDoneX += todoX;
  }

  hipDeviceSynchronize();
}


void findRangeWrap(matrix dD, real *dranges, int cntWant){
  dim3 block(4*BLOCK_SIZE,BLOCK_SIZE/4);
  dim3 grid(1,4*(dD.pr/BLOCK_SIZE));

  findRangeKernel<<<grid,block>>>(dD,dranges,cntWant);

  
  hipDeviceSynchronize();
}

void rangeSearchWrap(matrix dD, real *dranges, charMatrix dir){
  dim3 block(BLOCK_SIZE,BLOCK_SIZE);
  dim3 grid(dD.pc/BLOCK_SIZE,dD.pr/BLOCK_SIZE);

  int todoX, todoY, numDoneX, numDoneY;
  
  numDoneX = 0;
  while ( numDoneX < dD.pc ){
    todoX = min( dD.pc - numDoneX, MAX_BS*BLOCK_SIZE );
    grid.x = todoX/BLOCK_SIZE;
    numDoneY = 0;
    while( numDoneY < dD.pr ){
      todoY = min( dD.pr - numDoneY, MAX_BS*BLOCK_SIZE );
      grid.y = todoY/BLOCK_SIZE;
      rangeSearchKernel<<<grid,block>>>(dD, numDoneX, numDoneY, dranges, dir);
      numDoneY += todoY;
    }
    numDoneX += todoX;
  }

  hipDeviceSynchronize();
}

void nnWrap(const matrix dx, const matrix dy, real *dMins, int *dMinIDs){
  dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
  dim3 dimGrid;
  
  dimGrid.x = 1;
  dimGrid.y = dx.pr/dimBlock.y + (dx.pr%dimBlock.y==0 ? 0 : 1);
  nnKernel<<<dimGrid,dimBlock>>>(dx,dy,dMins,dMinIDs);
  hipDeviceSynchronize();
}


void rangeCountWrap(const matrix dq, const matrix dx, real *dranges, int *dcounts){
  dim3 block(BLOCK_SIZE,BLOCK_SIZE);
  dim3 grid(1,dq.pr/BLOCK_SIZE);

  rangeCountKernel<<<grid,block>>>(dq,dx,dranges,dcounts);
  hipDeviceSynchronize();
}


/*NOTE: can be deleted */
void pruneWrap(charMatrix dcM, matrix dD, real *dradiiX, real *dradiiQ){
  dim3 block(BLOCK_SIZE,BLOCK_SIZE);
  dim3 grid(dD.pr/BLOCK_SIZE,dD.pc/BLOCK_SIZE);
  
  pruneKernel<<<grid,block>>>(dD,dradiiX,dradiiQ,dcM);
  hipDeviceSynchronize();
}
#endif
